#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <thrust/device_vector.h>

using namespace std;

#define ERROR_TOL 0.0001


__host__ int checkIfDD(int numUnknowns, float mat[])
{
	int   diagCounter=0, offDiagCounter=0, dd=0;
	float chkdd=0;
	float sumdd=0;
	//int *nonDDRows;

	//nonDDRows = (int *)malloc(numUnknowns * sizeof(int *));

	printf("\nChecking if the matrix is (strictly) diagonally dominant...\n");

	//ROW-WISE diagonally dominant check 
	for (diagCounter = 0; diagCounter < numUnknowns*numUnknowns; diagCounter =  diagCounter + numUnknowns+1)
	{ 
		chkdd = 0;
		chkdd = mat[diagCounter];//diagonal terms
		//printf("chkdd [diagCounter] %2d %i \n", chkdd, diagCounter);
		sumdd = 0;
		offDiagCounter = 1;
		for (offDiagCounter = diagCounter; offDiagCounter < (diagCounter + numUnknowns) && offDiagCounter <= (numUnknowns*numUnknowns); offDiagCounter = offDiagCounter + 1)
		{
			//printf(" in loop sumdd [offDiagCounter] %2d %i \n", sumdd, offDiagCounter);
			if (offDiagCounter != diagCounter){ 
				sumdd = sumdd + mat[offDiagCounter]; }
			//printf(" in if statement sumdd [offDiagCounter] %2d %i \n", sumdd, offDiagCounter);
		}
			if (sumdd <= chkdd){ dd++; }
	}
	if (dd == numUnknowns){
		printf(
			"\nYES ..."
			"\nThe matrix is (strictly) diagonally dominant.");
	}
	else{
		printf(
			"\nThe matrix is NOT (strictly) diagonally dominant.\nOnly %i rows are DD.\n",dd);
		return 0; /* false */
	}
	return 1; /* true */
}

__global__ void add(float *a_d,
	float *b_d,
	float * c_d,
	int dim)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < dim)
	{
		c_d[tid] = a_d[tid] + b_d[tid];
		tid += gridDim.x * blockDim.x;
	}
}

__global__ void substract(float *a_d,
	float *b_d,
	float *c_d,
	int dim)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < dim)
	{
		c_d[tid] = a_d[tid] - b_d[tid];
		tid += gridDim.x * blockDim.x;
	}
}

// this function is specifically made for the jacobi iterations.
// it will multiple vec[i] by frac{1}{diag[i]}

__global__ void diaMultVec(float * diag, float * vec, int dim)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < dim)
	{
		if (diag[tid] != 0)
		{
			vec[tid] /= diag[tid];
		}
		tid += gridDim.x * blockDim.x;
	}

}

// VecAbs() is the function to make Vec only contain the absolute value

__global__ void VecAbs(float * vec, int dim)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < dim)
	{
		if (vec[tid] < 0)
		{
			vec[tid] = -vec[tid];
		}
		tid += gridDim.x * blockDim.x;
	}

}
__host__ int getMatDim(FILE *fpMatA){
	int Matrix_Size[2];
	int NoofRows_A = 0, NoofCols_A = 0;
	fscanf(fpMatA, "%d %d\n", &NoofRows_A, &NoofCols_A);
	Matrix_Size[0] = NoofRows_A;
	Matrix_Size[1] = NoofCols_A;
	return Matrix_Size[0];
}

__host__ float** getVector(FILE *fpVectB, int sysDim)
{
	int irow = 0;
	float ** vectB;
	//vectB = new double[sysDim];
	vectB = (float **)malloc(sysDim * sizeof(float*));
		for (irow = 1; irow < sysDim+1; irow++){
			fscanf(fpVectB, "%f", &vectB[irow]);
	}
		return vectB;
}
// Make the VecMax just step by step to avoid __syncblocks()

__global__ void SwapForOddDim(float * vec, int dim)
{
	if (vec[0] < vec[dim - 1])
		vec[0] = vec[dim - 1];
}


__global__ void VecMaxOneStepCompare(float * vec, int dim)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int mid = dim / 2; // get the half size

	while (tid < mid)
	{
		if (tid < mid)    // filter the active thread
		{
			if (vec[tid] < vec[tid + mid]) // get the larger one between vec[tid] and vec[tid+mid]
				vec[tid] = vec[tid + mid];  // and store the larger one in vec[tid]
		}

		tid += gridDim.x * blockDim.x;
	}
	__syncthreads();
}
	
// VecMax() is the norm of a vector. Since we are using infinity norm
// we just call it VecMax instead of VecNorm.
// dim:  the size vector
// the max value is stored at vec[0]
// the content of vec will be changed

void VecMax(float * vec, int dim, int dimB, int dimT)
{

	while (dim > 1)
	{
		VecMaxOneStepCompare << <dimB, dimT >> >(vec, dim);

		// if the dimension is odd.
		if (dim % 2)
		{
			SwapForOddDim << <1, 1 >> >(vec, dim);
		}
		dim /= 2;        // make the vector half size short.
	}

	// compare last two values
	SwapForOddDim << <1, 1 >> >(vec, 2);
}



// The following fucntion does the job of matrix-vector multiplication.
// Well, in this homework, although we can assume the matrix is square,
// the matMultVec is written for a more general case.
// 
// mat_A:  A dim_row by dim_col matrix which is stored in row-major manner.
// vec:    A dim_col dimension vector.
// rst:    The result dim_row dimension vector.

__global__ void matMultVec(float * mat_A,
	float * vec,
	float * rst,
	int dim_row,
	int dim_col)
{
	int rowIdx = threadIdx.x + blockIdx.x * blockDim.x; // Get the row Index 
	int aIdx;
	while (rowIdx < dim_row)
	{
		rst[rowIdx] = 0; // clean the value at first
		for (int i = 0; i < dim_col; i++)
		{
			aIdx = rowIdx * dim_col + i; // Get the index for the element a_{rowIdx, i}
			rst[rowIdx] += (mat_A[aIdx] * vec[i]); // do the multiplication
		}
		rowIdx += gridDim.x * blockDim.x;
	}
	__syncthreads();
}


/*******************************
function jacobi (GPU Version):
The following function does the main things.

A X = B

A: the dim by dim matrix
B: the dim dimenstion vector
X: the solution

*******************************/

void jacobi(float * A, float * B, float * X, int dim)
{
	float * diag;   // 1D array: a vector with diagonal elements from A
	float * LU;     // 1D array: a matrix with off-diagonal elements from A, row major(?)
	float * x_old;

	diag = new float[dim];
	LU = new float[dim*dim];
	x_old = new float[dim];

	// initialize diag, LU
	for (int i = 0; i < dim; i++)
	{
		diag[i] = A[i*dim + i];
		for (int j = 0; j < dim; j++)
		{
			LU[i*dim + j] = A[i*dim + j];
		}
		LU[i*dim + i] = 0;
	}

	// initialize the block dimension and thread dimension.
	// find the best dimB and dimT to fit the dim
	int dimB, dimT;
	dimT = 8; //originally 256
	dimB = (dim / dimT) + 1;
	printf("\nThread size = %i, # of Blocks = %i \n", dimT, dimB);

	float err = 1.0, checkDotSum = 0;

	// set up the memory for GPU
	float * LU_d;
	float * B_d;
	float * diag_d;
	float *X_d, *X_old_d;
	float * tmp;
	float * tmp_h;

	tmp_h = (float *)malloc(dim * sizeof(float *)); //for final dot product check
	hipMalloc((void **)&B_d, sizeof(float) * dim);
	hipMalloc((void **)&diag_d, sizeof(float) * dim);
	hipMalloc((void **)&LU_d, sizeof(float) * dim * dim);

	hipMemcpy(LU_d, LU, sizeof(float) * dim * dim, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, sizeof(float) * dim, hipMemcpyHostToDevice);
	hipMemcpy(diag_d, diag, sizeof(float) * dim, hipMemcpyHostToDevice);

	hipMalloc((void **)&X_d, sizeof(float) * dim);
	hipMalloc((void **)&X_old_d, sizeof(float) * dim);
	hipMalloc((void **)&tmp, sizeof(float) * dim);

	float * max;
	max = new float;
	int count = 0;
	while (err > ERROR_TOL) // do the iteration untill err is less than tolerance
	{
		count++;
		// 1. Copy X to x_old
		for (int i = 0; i < dim; i++){
			x_old[i] = X[i];
		}
		// 2. Compute X by A x_old

		hipMemcpy(X_old_d, x_old, sizeof(float) * dim, hipMemcpyHostToDevice);
		matMultVec << <dimB, dimT >> >(LU_d, X_old_d, tmp, dim, dim); // use x_old to compute LU X_old and store the result in tmp
		substract << <dimB, dimT >> >(B_d, tmp, X_d, dim);        // get the (B - LU X_old), which is stored in X_d
		diaMultVec << <dimB, dimT >> >(diag_d, X_d, dim);         // get the new X

		// 3. copy the new X back to the Host Memory
		hipMemcpy(X, X_d, sizeof(float) * dim, hipMemcpyDeviceToHost);

		// 4. calculate the norm of X_new - X_old
		substract << <dimB, dimT >> >(X_old_d, X_d, tmp, dim);
		VecAbs << <dimB, dimT >> >(tmp, dim);
		VecMax(tmp, dim, dimB, dimT);

		// copy the max value from Device to Host

		hipMemcpy(max, tmp, sizeof(float), hipMemcpyDeviceToHost);
		err = (*max);
		printf("\nIteration #%i, Error: %f", count, err);
	}

	cout<<"\n\nThis converged after "<<count<<" iterations.\n \n";

	//Check for correctness
	//initialize tmp_h 
	for (int i = 0; i < dim; i++){
		tmp_h[i] = 0;
	}
	matMultVec << <dimB, dimT >> >(A, X, tmp_h, dim, dim); //tmp stores Ax
	for (int i = 0; i < dim; i++){
		checkDotSum += tmp_h[i] * B[i];
	}//check (Ax) dot B 
	printf("Check: [Ax] dot [B] = %f \n \n", checkDotSum);

	// free memory after all iterations
	hipFree(LU_d);
	hipFree(B_d);
	hipFree(diag_d);
	hipFree(tmp);
	hipFree(X_old_d);
	hipFree(X_d);
	delete[] diag;
	delete[] LU;
	delete[] x_old;
	delete max;

}

double test(int n)
{
	//double **mat;
	//float * A; //probably needs to be deleted. 
	//float * B;
	float * vectX; 
	float *vectB;
	float *matA;
	int vectDim=0;
	float initialScalingFactor = 1; 
	
	FILE *fpM;
	fpM = fopen("H:\\UI Comp Sci Work\\Parallel\\Assignment1\\MatrixDD.txt", "r");//DOUBLE SLASH FOR ADDRESSES
	printf("\nGetting system dimension...");
	int dim = getMatDim(fpM);
	printf("\nSystem dimension is %i.\n", dim);
	
	int irow = 0;
	matA = new float[dim*dim];
	for (irow = 0; irow < dim * dim + 1; irow++){
		//if (irow > 1){ fscanf(fpM, "%f\n", &matA[irow]); } //ROW MAJOR
		fscanf(fpM, "%f\n", &matA[irow]);
		//printf("%f\n",matA[irow]);
	}
	fclose(fpM);
	printf("\nSuccessfully read the matrix file.");

	checkIfDD(dim, matA);

	FILE *fpV;
	fpV = fopen("H:\\UI Comp Sci Work\\Parallel\\Assignment1\\RandomVector.txt", "r");

	fscanf(fpV, "%f\n", &vectDim);//checks if vectDim is the same dimension

	vectB = new float [dim];
	vectX = new float[dim];
	irow = 0;
	for (irow = 0; irow < dim + 1; irow++){
		//if (irow > 1){fscanf(fpV, "%f\n", &vectB[irow]); }
		fscanf(fpV, "%f\n", &vectB[irow]);
		vectX[irow] = initialScalingFactor * vectB[irow]; //initial guess
		//printf("%f\n",vectB[irow]);
	}
	printf("\nSuccessfully read the vector file.");
	printf("\nInitialized x to be %.2f * b.",initialScalingFactor);
	//getVector(fpV, dim);

	fclose(fpV);

	//get the time
	clock_t start, finish;
	double totaltime;
	start = clock();

	printf("\n\nRunning Jacobi iteration...");
	jacobi(matA, vectB, vectX, dim);  // do the jacobi

	finish = clock();
	totaltime = (double)(finish - start) / CLOCKS_PER_SEC;

	//if (c == 'y')
	//{
	//	for (int i = 0; i < dim; i++)
	//	{
	//		cout << vectX[i] << endl;
	//	}
	//}

	FILE *fpR; //result vector
	fpR = fopen("H:\\UI Comp Sci Work\\Parallel\\Assignment 3\\JacobiResultVector.txt", "w+");//double check path. 
	if (fpR != NULL){
		for (irow = 0; irow < dim; irow++){
			fprintf(fpR,"%f\n",vectX[irow]);
		}
	}
	fclose(fpR);

	FILE *fpRP; //performance results
	fpRP = fopen("H:\\UI Comp Sci Work\\Parallel\\Assignment 3\\PerformanceResults.txt", "w+");//double check path. 
	if (fpRP != NULL){
			fprintf(fpRP, "%d\n", totaltime);
	}
	else printf("Can't write the result vector. Check the path or filename");
	

	delete[] matA;
	delete[] vectB;
	delete[] vectX;
	return totaltime;
}


int main(int argc, char *argv[])
{
	
	int size = 2048;
	double t = test(size);
	cout << "Wall time:" << t << " secs" << endl;
	printf("\nSee JacobiResultVector.txt for X.");
	return 0;
}

